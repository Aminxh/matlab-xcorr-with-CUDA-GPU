#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include ""
#include <iostream>
#include <numeric>
#include <vector>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <cmath>
#include "xcorr_cuda.h"
#include <algorithm>
#include <mkl/mkl.h>
#include <complex>

using namespace std;

#define PI 3.141592653589
#define BLOCKSIZE 256

__global__ void shiftFreqSig_Kernel(hipfftComplex* shiftedSig,  hipfftComplex* mainSig, int fd, float fs, int64_t sigSize)
{
    int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < sigSize)
    {
        float angle = 2.0f * PI * fd * (idx + 1.0f)/fs;
        float realPart, imagPart;
        __sincosf(angle, &imagPart, &realPart);

        hipfftComplex temp = mainSig[idx];
        hipfftComplex calcVal;
        calcVal.x = (temp.x*realPart - temp.y*imagPart);
        calcVal.y = (temp.x*imagPart + temp.y*realPart);

        shiftedSig[idx] = calcVal;
    }
}

__global__ void multipleSigs_conj( hipfftComplex* __restrict__ X,  hipfftComplex* __restrict__ Y, hipfftComplex* __restrict__ R, int sigSize)
{
    int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(idx < sigSize)
    {
        hipfftComplex x = X[idx];
        hipfftComplex y = Y[idx];

        hipfftComplex multiple;
        multiple.x = x.x * y.x + x.y * y.y;
        multiple.y = - x.x * y.y + x.y * y.x;

        R[idx] = multiple;
    }
}

__global__ void ifftNomalizer_Kernel(hipfftComplex* sig, int sigSize, float invVal)
{
    int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(idx < sigSize)
    {
        sig[idx].x *= invVal;
        sig[idx].y *= invVal;
    }
}

__global__ void fftShift( hipfftComplex* sig, hipfftComplex* shiftedSig, int sigSize)
{
    int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(idx < sigSize)
    {
        int halfSize = (sigSize+1) >> 1;
        int shifted_idx = (idx + halfSize) % sigSize;
        shiftedSig[idx] = sig[shifted_idx];
    }
}

__global__ void abs_cuFFT(const hipfftComplex* sig, const int64_t* index_range, float* output, int indexRangeSize)
{
    int64_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < indexRangeSize)
    {
        int idx = index_range[i];
        hipfftComplex val = sig[idx];
        output[i] = hypotf(val.x, val.y);
    }

}

//-----------------------------------------------
//
//
// Notice : change local variables in xcorr function yourself
//
//
//----------------------------------------------

 vector<vector<float>> xcorr(hipfftComplex* primSig, hipfftComplex* secondSig, int64_t lenPrimSig, int64_t lenSecondSig)
{

    auto start = chrono::high_resolution_clock::now();

    // fd initialization ...
    float fs = 500e3;

    // leakage initialization ...
    double leakage_time = 100e-6;
    double leakage_freq = 50;

    // Res_freq_initial initialization ...
    float Res_freq_initial = 1;

    // fd initialization ...
    int fd_lowerBound = -10000;
    int fd_upperBound =  10000;
    int fd_range = ceil((double)((fd_upperBound - fd_lowerBound) + 1)/Res_freq_initial);
    int *fd = new int[fd_range];

    for (int i = 0; i < fd_range ; i++)
        fd[i] = fd_lowerBound + i*Res_freq_initial;

    // approx_delay , lag_center, lag_radius and lags initialization ...
    double approx_delay = 0.003;
    double lag_center = round(approx_delay * fs);
    double lag_radius = abs(lag_center);
    int *lags = new int[2 * (int64_t)lag_radius + 1];

    for (int i = 0; i < 2 * lag_radius + 1; ++i)
        lags[i] = -1 * lag_radius + i;

    // defining Amb and initializating with 0 ...
    vector<vector<float>> Amb(fd_range, vector<float>(2 * lag_radius + 1, 0.0));

    // Performing fft on X in cuda ...
    hipfftComplex *d_input, *d_X, *X;
    X = new hipfftComplex[lenPrimSig];

    hipMalloc((void **)&d_input, sizeof(hipfftComplex) * lenPrimSig);
    hipMalloc((void **)&d_X, sizeof(hipfftComplex) * lenPrimSig);
    hipMemcpy(d_input, primSig, sizeof(hipfftComplex) * lenPrimSig, hipMemcpyHostToDevice);

    hipfftHandle plan_X;
    hipfftPlan1d(&plan_X, lenPrimSig, HIPFFT_C2C, 1);
    hipfftExecC2C(plan_X, d_input, d_X, HIPFFT_FORWARD);

    hipMemcpy(X, d_X, sizeof(hipfftComplex) * lenPrimSig, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipfftDestroy(plan_X);

    // Copy secondSig to GPU to prevent mutiple copying in for loop ...
    hipfftComplex *d_secondSig;
    hipMalloc((void **)&d_secondSig, sizeof(hipfftComplex) * lenPrimSig);
    hipMemcpy(d_secondSig, secondSig, sizeof(hipfftComplex) * lenPrimSig, hipMemcpyHostToDevice);

    // Calculating shifting signal on GPU ...
    hipfftComplex *d_shiftedSig;
    hipMalloc((void **)&d_shiftedSig, sizeof(hipfftComplex) * lenPrimSig);

    // Calculating fft of shiftedSig and puting result in d_Y ...
    hipfftComplex *d_Y;
    hipMalloc((void **)&d_Y, sizeof(hipfftComplex) * lenPrimSig);

    hipfftHandle plan;
    hipfftPlan1d(&plan, lenPrimSig, HIPFFT_C2C, 1);

    // Calculating X .* Conj(Y) and puting result in d_R
    hipfftComplex *d_R;
    hipMalloc((void **)&d_R, lenPrimSig * sizeof(hipfftComplex));

    // Calculatin ifft(X .* Conj(Y)) and puting result to d_final_R ...
    hipfftComplex *d_final_R;
    hipMalloc((void **)&d_final_R, sizeof(hipfftComplex) * lenPrimSig);

    hipfftHandle ifft_plan;
    hipfftPlan1d(&ifft_plan, lenPrimSig , HIPFFT_C2C, 1);

    // Defining center, index_range and intializing them ...
    int64_t center = floor(lenPrimSig/2) + 1;
    int64_t *index_range = new int64_t[2 * (int64_t)lag_radius + 1];

    for(int i = 0 ; i < 2 * (int64_t)lag_radius + 1 ; i++)
        index_range[i] = lags[i] + center - 1;

    // Calculating fftShift(d_final_R) and put the result to d_shiftedFFTSig ...
    hipfftComplex *d_shiftedFFTSig;
    hipMalloc((void **)&d_shiftedFFTSig, sizeof(hipfftComplex) * lenPrimSig);

    // Copy the index_range to Calculate abs(d_shiftedFFTSig(index_range)) ...
    int64_t *d_index_range;
    float* d_amb_sig;

    hipMalloc((void **)&d_index_range, (2 * (int64_t)lag_radius + 1) * sizeof(int64_t));
    hipMalloc((void **)&d_amb_sig, (2 * (int64_t)lag_radius + 1) * sizeof(float));
    hipMemcpy(d_index_range, index_range, (2 * (int64_t)lag_radius + 1) * sizeof(int64_t), hipMemcpyHostToDevice);

    for (int i = 0 ; i < fd_range; ++i)
    {

        shiftFreqSig_Kernel<<<(lenPrimSig + BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_shiftedSig, d_secondSig, fd[i], fs, lenPrimSig);

        hipfftExecC2C(plan, d_shiftedSig, d_Y, HIPFFT_FORWARD);
        hipDeviceSynchronize();

        multipleSigs_conj<<<(lenPrimSig + BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_X, d_Y, d_R, lenPrimSig);

        hipfftExecC2C(ifft_plan, d_R, d_final_R, HIPFFT_BACKWARD);

        // Normalizing ifft , because ifft in cuda is unnormalized ...
        ifftNomalizer_Kernel<<<(lenPrimSig + BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_final_R, lenPrimSig, 1.0f/lenPrimSig);

        fftShift<<<(lenPrimSig + BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_final_R, d_shiftedFFTSig, lenPrimSig);

        abs_cuFFT<<<(2 * (int64_t)lag_radius + 1 + BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_shiftedFFTSig, d_index_range, d_amb_sig, (2 * (int64_t)lag_radius + 1));

        hipMemcpy(Amb[i].data(), d_amb_sig, (2 * (int64_t)lag_radius + 1) * sizeof(float), hipMemcpyDeviceToHost);

    }

    // Free allocated memories on GPU ...
    hipFree(d_R);
    hipFree(d_Y);
    hipFree(d_amb_sig);
    hipFree(d_final_R);
    hipFree(d_index_range);
    hipFree(d_shiftedFFTSig);
    hipFree(d_shiftedSig);
    hipFree(d_X);
    hipFree(d_secondSig);
    hipfftDestroy(ifft_plan);
    hipfftDestroy(plan);
    delete [] index_range;

    // returning output, you can also adjust this part yourself , like sending by reffrence , pointer and ...
    return Amb;
   }
